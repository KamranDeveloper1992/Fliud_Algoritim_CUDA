#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void SimulationFliud(int sizeFluid, int rateFluid, int* status) {
    if (sizeFluid > 50 || rateFluid < 20) {
        *status = 1;  
    }
    else {
        *status = 0; 
    }
}
int main() {
    int result = -1;
    int* d_result;

    hipMalloc((void**)&d_result, sizeof(int));


    SimulationFliud << <1, 1 >> > (100, 200, d_result);


    hipFree(d_result);

    hipDeviceReset();

    return 0;
}
